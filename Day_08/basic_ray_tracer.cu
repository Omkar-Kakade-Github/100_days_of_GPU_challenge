#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include "cpu_bitmap.h"

#define INF     2e10f
#define DIM 1024
#define rnd( x ) (x * rand() / RAND_MAX)
#define SPHERES 20

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


struct Sphere {
    float r, g, b;
    float radius;
    float x,y,z;
    __device__ float hit ( float ox, float oy, float *n) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy);
            *n = dz / sqrtf( radius*radius );
            return dz + z;
        }
        return -INF;
    }
};

__constant__ Sphere s[SPHERES];

__global__ void kernel( unsigned char *ptr ) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    float   ox = (x - DIM/2);
    float   oy = (y - DIM/2);

    float r=0, g=0, b=0;
    float max_z = -INF;
    for(int i=0; i<SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if(t > max_z) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            max_z = t;
        }
    }

    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;

}

struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main(void) {
    // DataBlock   data;

    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    CPUBitmap bitmap( DIM, DIM);
    unsigned char *dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap,
                              bitmap.image_size() ) );

    Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
    for (int i=0; i<SPHERES; i++) {
        temp_s[i].r = rnd( 1.0f );
        temp_s[i].g = rnd( 1.0f );
        temp_s[i].b = rnd( 1.0f );
        temp_s[i].x = rnd( 1000.0f ) - 500;
        temp_s[i].y = rnd( 1000.0f ) - 500;
        temp_s[i].z = rnd( 1000.0f ) - 500;
        temp_s[i].radius = rnd( 100.0f ) + 20;
    }

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( s), temp_s, 
                                sizeof(Sphere) * SPHERES) );

    free(temp_s);

    dim3 grids(DIM/16,DIM/16);
    dim3 threads(16,16);
    kernel<<<grids,threads>>>(dev_bitmap);

    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    HANDLE_ERROR( hipFree( dev_bitmap ) );

    bitmap.display_and_exit();
}
