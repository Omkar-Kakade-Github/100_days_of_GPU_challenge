#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32

__global__ void matmul_cuda(float *A, float *B, float *C, int N) {
    __shared__ float sA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sB[BLOCK_SIZE][BLOCK_SIZE];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * BLOCK_SIZE + ty;
    int col = blockIdx.x * BLOCK_SIZE + tx;
    
    float value = 0.0f;
    
    for (int i = 0; i < N / BLOCK_SIZE; ++i) {
        sA[ty][tx] = A[row * N + (i * BLOCK_SIZE + tx)];
        sB[ty][tx] = B[(i * BLOCK_SIZE + ty) * N + col];
        
        __syncthreads();
        
        #pragma unroll
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            value += sA[ty][j] * sB[j][tx];
        }
        
        __syncthreads();
    }
    
    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

extern "C" void run_cuda(float *h_A, float *h_B, float *h_C, int N, float *elapsed_time) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matmul_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsed_time, start, stop);
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
